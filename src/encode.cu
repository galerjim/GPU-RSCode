#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  encode.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/2012 10:42:32 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Shuai YUAN (yszheda AT gmail.com), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include "matrix.h"

void write_metadata(int totalSize, int parityBlockNum, int nativeBlockNum, uint8_t *encodingMatrix)
{
	FILE *fp;
	if( ( fp = fopen(".METADATA", "wb") ) == NULL )
	{
		printf("Can not open META file!\n");
		exit(0);
	}
	fprintf(fp, "%d\n", totalSize);
	fprintf(fp, "%d %d\n", parityBlockNum, nativeBlockNum);
	int i;
	int j;
	for(i=0; i<nativeBlockNum; i++)
	{
		for(j=0; j<nativeBlockNum; j++)
		{
			if(i == j)
			{
				fprintf(fp, "1 ");
			}
			else
			{
				fprintf(fp, "0 ");
			}
		}
		fprintf(fp, "\n");
	}
	for(i=0; i<parityBlockNum; i++)
	{
		for(j=0; j<nativeBlockNum; j++)
		{
			fprintf(fp, "%d ", encodingMatrix[i*nativeBlockNum+j]);
		}
		fprintf(fp, "\n");
	}
	fclose(fp);
}

extern "C"
void encode(uint8_t *dataBuf, uint8_t *codeBuf, int nativeBlockNum, int parityBlockNum, int chunkSize, int totalSize)
{
	uint8_t *dataBuf_d;		//device
	uint8_t *codeBuf_d;		//device
	int dataSize = nativeBlockNum*chunkSize*sizeof(uint8_t);
	int codeSize = parityBlockNum*chunkSize*sizeof(uint8_t);
	hipMalloc( (void **)&dataBuf_d, nativeBlockNum*chunkSize*sizeof(uint8_t) );
//	hipMemset(dataBuf_d, 0, dataSize);
	hipMalloc( (void **)&codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t) );
//	hipMemset(codeBuf_d, 0, codeSize);

	hipMemcpy(dataBuf_d, dataBuf, nativeBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyHostToDevice);

	uint8_t *encodingMatrix;	//host
	uint8_t *encodingMatrix_d;	//device
	encodingMatrix = (uint8_t*) malloc( parityBlockNum*nativeBlockNum*sizeof(uint8_t) );
	hipMalloc( (void **)&encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t) );


	float time;
	// compute the execution time
	hipEvent_t start, stop;
	// create event
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// record event
	hipEventRecord(start);
	dim3 blk(parityBlockNum, nativeBlockNum);
	gen_encoding_matrix<<<1, blk>>>(encodingMatrix_d, parityBlockNum, nativeBlockNum);
//	hipDeviceSynchronize();
	// record event and synchronize
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	// get event elapsed time
	hipEventElapsedTime(&time, start, stop);
	printf("Generating encoding matrix completed: %fms\n", time);

	hipMemcpy(encodingMatrix, encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t), hipMemcpyDeviceToHost);
	write_metadata(totalSize, parityBlockNum, nativeBlockNum, encodingMatrix);

	// record event
	hipEventRecord(start);
//	int gridDimX = (int)(ceil((float)chunkSize/TILE_WIDTH));
//	int gridDimY = (int)(ceil((float)parityBlockNum/TILE_WIDTH));
//	dim3 grid(gridDimX, gridDimY);
//	dim3 block(TILE_WIDTH, TILE_WIDTH);
	int gridDimX = min( (int)( ceil((float)chunkSize / TILE_WIDTH_COL) ), SINGLE_GRID_SIZE );
	int gridDimY = (int)( ceil((float)nativeBlockNum / TILE_WIDTH_ROW) );
	dim3 grid(gridDimX, gridDimY);
//	dim3 block(TILE_WIDTH_ROW, TILE_WIDTH_COL);
	dim3 block(TILE_WIDTH_COL, TILE_WIDTH_ROW);
	encode_chunk<<<grid, block>>>(dataBuf_d, encodingMatrix_d, codeBuf_d, nativeBlockNum, parityBlockNum, chunkSize);
	// record event and synchronize
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	// get event elapsed time
	hipEventElapsedTime(&time, start, stop);
	printf("Encoding file completed: %fms\n", time);

	hipMemcpy(codeBuf, codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(encodingMatrix_d);
	hipFree(dataBuf_d);
	hipFree(codeBuf_d);

	free(encodingMatrix);

}

extern "C"
void encode_file(char *file, int nativeBlockNum, int parityBlockNum)
{
	int chunkSize = 1;
	int totalSize;

	FILE *fp_in;
	FILE *fp_out;
	if( ( fp_in = fopen(file,"rb") ) == NULL )
	{
		printf("Can not open source file!\n");
		exit(0);
	}

	fseek(fp_in, 0L, SEEK_END);
	//ftell() get the total size of the file
	totalSize = ftell(fp_in);
	chunkSize = (ftell(fp_in) / nativeBlockNum) + ( ftell(fp_in)%nativeBlockNum != 0 ); 
//	chunkSize = (int) (ceil( (long double) (ftell(fp_in) / nativeBlockNum)) ); 

	uint8_t *dataBuf;		//host
	uint8_t *codeBuf;		//host
	int dataSize = nativeBlockNum*chunkSize*sizeof(uint8_t);
	int codeSize = parityBlockNum*chunkSize*sizeof(uint8_t);
	dataBuf = (uint8_t*) malloc( nativeBlockNum*chunkSize*sizeof(uint8_t) );
	memset(dataBuf, 0, dataSize);
	codeBuf = (uint8_t*) malloc( parityBlockNum*chunkSize*sizeof(uint8_t) );
	memset(codeBuf, 0, codeSize);
	
	int i;
	for(i=0; i<nativeBlockNum; i++)
	{
		if( fseek( fp_in, i*chunkSize, SEEK_SET ) == -1 )
		{
			printf("fseek error!\n");
			exit(0);
		}

		if( fread( dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_in ) == EOF )
		{
			printf("fread error!\n");
			exit(0);
		}
	}
	fclose(fp_in);
	
	encode(dataBuf, codeBuf, nativeBlockNum, parityBlockNum, chunkSize, totalSize);

	char output_file_name[100];
	for(i=0; i<nativeBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_", i);
		strcat(output_file_name, file);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open source file!\n");
			exit(0);
		}
		if( fwrite(dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}
	for(i=0; i<parityBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_", i+nativeBlockNum);
		strcat(output_file_name, file);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open source file!\n");
			exit(0);
		}
		if( fwrite(codeBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}

	free(dataBuf);
	free(codeBuf);
}
