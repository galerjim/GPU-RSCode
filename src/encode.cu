#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  encode.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/2012 10:42:32 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Shuai YUAN (yszheda AT gmail.com), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include "matrix.h"

void write_metadata(char *fileName, int totalSize, int parityBlockNum, int nativeBlockNum, uint8_t* encodingMatrix)
{
	FILE *fp;
	if( ( fp = fopen(fileName, "wb") ) == NULL )
	{
		printf("Can not open META file!\n");
		exit(0);
	}
	fprintf(fp, "%d\n", totalSize);
	fprintf(fp, "%d %d\n", parityBlockNum, nativeBlockNum);
	for (int i = 0; i < nativeBlockNum; ++i)
	{
		for (int j = 0; j < nativeBlockNum; ++j)
		{
			if (i == j)
			{
				fprintf(fp, "1 ");
			}
			else
			{
				fprintf(fp, "0 ");
			}
		}
		fprintf(fp, "\n");
	}
	for (int i = 0; i < parityBlockNum; ++i)
	{
		for (int j = 0; j < nativeBlockNum; ++j)
		{
			fprintf(fp, "%d ", encodingMatrix[i*nativeBlockNum + j]);
		}
		fprintf(fp, "\n");
	}
	fclose(fp);
}

extern "C"
void encode(char *fileName, uint8_t *dataBuf, uint8_t *codeBuf, int nativeBlockNum, int parityBlockNum, int chunkSize, int totalSize)
{
	uint8_t *dataBuf_d;		//device
	uint8_t *codeBuf_d;		//device
	int dataSize = nativeBlockNum*chunkSize*sizeof(uint8_t);
	int codeSize = parityBlockNum*chunkSize*sizeof(uint8_t);

	float totalComputationTime = 0;
	float totalCommunicationTime = 0;
	// compute total execution time
	float totalTime;
	hipEvent_t totalStart, totalStop;
	// create event
	hipEventCreate(&totalStart);
	hipEventCreate(&totalStop);
	hipEventRecord(totalStart);

	hipMalloc( (void **)&dataBuf_d, nativeBlockNum*chunkSize*sizeof(uint8_t) );
//	hipMemset(dataBuf_d, 0, dataSize);
	hipMalloc( (void **)&codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t) );
//	hipMemset(codeBuf_d, 0, codeSize);

	// compute step execution time
	float stepTime;
	hipEvent_t stepStart, stepStop;
	// create event
	hipEventCreate(&stepStart);
	hipEventCreate(&stepStop);

	// record event
	hipEventRecord(stepStart);
	hipMemcpy(dataBuf_d, dataBuf, nativeBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyHostToDevice);
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("Copy data from CPU to GPU: %fms\n", stepTime);
	totalCommunicationTime += stepTime;

	uint8_t *encodingMatrix;	//host
	uint8_t *encodingMatrix_d;	//device
	encodingMatrix = (uint8_t*) malloc( parityBlockNum*nativeBlockNum*sizeof(uint8_t) );
	hipMalloc( (void **)&encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t) );

	// record event
	hipEventRecord(stepStart);
	dim3 blk(parityBlockNum, nativeBlockNum);
	gen_encoding_matrix<<<1, blk>>>(encodingMatrix_d, parityBlockNum, nativeBlockNum);
//	hipDeviceSynchronize();
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("Generating encoding matrix completed: %fms\n", stepTime);
	totalComputationTime += stepTime;

	// record event
	hipEventRecord(stepStart);
	hipMemcpy(encodingMatrix, encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t), hipMemcpyDeviceToHost);
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("Copy encoding matrix from GPU to CPU: %fms\n", stepTime);
	totalCommunicationTime += stepTime;

	// TO-DO: better tiling
	int gridDimX = min( (int)( ceil((float)chunkSize / TILE_WIDTH_COL) ), SINGLE_GRID_SIZE );
	int gridDimY = (int)( ceil((float)nativeBlockNum / TILE_WIDTH_ROW) );
	dim3 grid(gridDimX, gridDimY);
	dim3 block(TILE_WIDTH_COL, TILE_WIDTH_ROW);
	// record event
	hipEventRecord(stepStart);
	encode_chunk<TILE_WIDTH_ROW, TILE_WIDTH_COL, TILE_DEPTH><<<grid, block>>>(dataBuf_d, encodingMatrix_d, codeBuf_d, nativeBlockNum, parityBlockNum, chunkSize);
//	matrix_mul<TILE_WIDTH_ROW, TILE_WIDTH_COL, TILE_DEPTH><grid, block>(encodingMatrix_d, dataBuf_d, codeBuf_d, parityBlockNum, nativeBlockNum, chunkSize);
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("Encoding file completed: %fms\n", stepTime);
	totalComputationTime += stepTime;

	// record event
	hipEventRecord(stepStart);
	hipMemcpy(codeBuf, codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyDeviceToHost);
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("copy code from GPU to CPU: %fms\n", stepTime);
	totalCommunicationTime += stepTime;

	hipFree(encodingMatrix_d);
	hipFree(dataBuf_d);
	hipFree(codeBuf_d);

	// record event and synchronize
	hipEventRecord(totalStop);
	hipEventSynchronize(totalStop);
	// get event elapsed time
	hipEventElapsedTime(&totalTime, totalStart, totalStop);
	printf("Total computation time: %fms\n", totalComputationTime);
	printf("Total communication time: %fms\n", totalCommunicationTime);
	printf("Total GPU encoding time: %fms\n", totalTime);

	char metadata_file_name[strlen(fileName) + 15];
	sprintf(metadata_file_name, "%s.METADATA", fileName);
	write_metadata(metadata_file_name, totalSize, parityBlockNum, nativeBlockNum, encodingMatrix);
	free(encodingMatrix);
}

extern "C"
void encode_file(char *fileName, int nativeBlockNum, int parityBlockNum)
{
	int chunkSize = 1;
	int totalSize;

	FILE *fp_in;
	FILE *fp_out;
	if( ( fp_in = fopen(fileName,"rb") ) == NULL )
	{
		printf("Can not open source file!\n");
		exit(0);
	}

	fseek(fp_in, 0L, SEEK_END);
	// ftell() get the total size of the file
	totalSize = ftell(fp_in);
	chunkSize = (totalSize / nativeBlockNum) + ( totalSize%nativeBlockNum != 0 ); 
//	chunkSize = (ftell(fp_in) / nativeBlockNum) + ( ftell(fp_in)%nativeBlockNum != 0 ); 
//	chunkSize = (int) (ceil( (long double) (ftell(fp_in) / nativeBlockNum)) ); 

	uint8_t *dataBuf;		//host
	uint8_t *codeBuf;		//host
	int dataSize = nativeBlockNum*chunkSize*sizeof(uint8_t);
	int codeSize = parityBlockNum*chunkSize*sizeof(uint8_t);
	dataBuf = (uint8_t*) malloc( nativeBlockNum*chunkSize*sizeof(uint8_t) );
	memset(dataBuf, 0, dataSize);
	codeBuf = (uint8_t*) malloc( parityBlockNum*chunkSize*sizeof(uint8_t) );
	memset(codeBuf, 0, codeSize);
	
	int i;
	for(i=0; i<nativeBlockNum; i++)
	{
		if( fseek( fp_in, i*chunkSize, SEEK_SET ) == -1 )
		{
			printf("fseek error!\n");
			exit(0);
		}

		if( fread( dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_in ) == EOF )
		{
			printf("fread error!\n");
			exit(0);
		}
	}
	fclose(fp_in);
	
	encode(fileName, dataBuf, codeBuf, nativeBlockNum, parityBlockNum, chunkSize, totalSize);

	char output_file_name[strlen(fileName) + 5];
	for(i=0; i<nativeBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_%s", i, fileName);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open output file!\n");
			exit(0);
		}
		if( fwrite(dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}
	for(i=0; i<parityBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_%s", i + nativeBlockNum, fileName);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open output file!\n");
			exit(0);
		}
		if( fwrite(codeBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}

	free(dataBuf);
	free(codeBuf);
}
