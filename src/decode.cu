#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  decode.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/2012 10:50:55 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Shuai YUAN (yszheda AT gmail.com), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <pthread.h>
#include "matrix.h"
#include "cpu-decode.h"
extern "C"		
void CPU_invert_matrix(uint8_t *matrix, uint8_t *result, int size);


// #define DEBUG

struct ThreadDataType {
	int id;
	int nativeBlockNum;
	int parityBlockNum;
	int chunkSize;
	int totalSize;
	uint8_t* dataBuf;
	uint8_t* codeBuf;
	uint8_t* decodingMatrix;
};	/* ----------  end of struct ThreadDataType  ---------- */

typedef struct ThreadDataType ThreadDataType;

static pthread_barrier_t barrier;

void show_squre_matrix(uint8_t *matrix, int size)
{
	for(int i = 0; i < size; i++)
	{
		for(int j = 0; j < size; j++)
		{
			printf("%d ", matrix[i*size+j]);
		}
		printf("\n");
	}
}

void copy_matrix(uint8_t *src, uint8_t *des, int srcRowIndex, int desRowIndex, int rowSize)
{
	for(int i = 0; i < rowSize; i++)
	{
		des[desRowIndex * rowSize + i] = src[srcRowIndex * rowSize + i];
	}
}

extern "C"
void decode(uint8_t *dataBuf, uint8_t *codeBuf, uint8_t *decodingMatrix, int id, int nativeBlockNum, int parityBlockNum, int chunkSize)
{
//	hipSetDevice(id);

	int dataSize = nativeBlockNum * chunkSize * sizeof(uint8_t);
	int codeSize = nativeBlockNum * chunkSize * sizeof(uint8_t);
	uint8_t *dataBuf_d;		//device
	uint8_t *codeBuf_d;		//device

	float totalComputationTime = 0;
	float totalCommunicationTime = 0;
	// compute total execution time
	float totalTime;
	hipEvent_t totalStart, totalStop;
	// create event
	hipEventCreate(&totalStart);
	hipEventCreate(&totalStop);
	hipEventRecord(totalStart);

	hipMalloc((void **) &dataBuf_d, dataSize);
//	hipMemset(dataBuf_d, 0, dataSize);
	hipMalloc((void **) &codeBuf_d, codeSize);
//	hipMemset(codeBuf_d, 0, codeSize);

	// compute step execution time
	float stepTime;
	hipEvent_t stepStart, stepStop;
	// create event
	hipEventCreate(&stepStart);
	hipEventCreate(&stepStop);

	// record event
	hipEventRecord(stepStart);
	hipMemcpy(codeBuf_d, codeBuf, codeSize, hipMemcpyHostToDevice);
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("Copy code from CPU to GPU: %fms\n", stepTime);
	totalCommunicationTime += stepTime;

	int matrixSize = nativeBlockNum * nativeBlockNum * sizeof(uint8_t);
//	uint8_t *encodingMatrix_d;	//device
	uint8_t *decodingMatrix_d;	//device
//	hipMalloc((void **) &encodingMatrix_d, matrixSize);
	hipMalloc((void **) &decodingMatrix_d, matrixSize);

	// record event
	hipEventRecord(stepStart);
	hipMemcpy(decodingMatrix_d, decodingMatrix, matrixSize, hipMemcpyHostToDevice);
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("Copy decoding matrix from CPU to GPU: %fms\n", stepTime);
	totalCommunicationTime += stepTime;

//	// record event
//	hipEventRecord(stepStart);
//	invert_matrix(encodingMatrix_d, decodingMatrix_d, nativeBlockNum);
//	// record event and synchronize
//	hipEventRecord(stepStop);
//	hipEventSynchronize(stepStop);
//	// get event elapsed time
//	hipEventElapsedTime(&stepTime, stepStart, stepStop);
//	printf("Generating decoding matrix completed: %fms\n", stepTime);
//	totalComputationTime += stepTime;
//
//#ifdef DEBUG
//	uint8_t *decodingMatrix;	//host
//	decodingMatrix = (uint8_t*) malloc(matrixSize);
//	hipMemcpy(decodingMatrix, decodingMatrix_d, matrixSize, hipMemcpyDeviceToHost);
//	show_squre_matrix(decodingMatrix, nativeBlockNum);
//	free(decodingMatrix);
//#endif
//
	stepTime = decode_chunk(dataBuf_d, decodingMatrix_d, codeBuf_d, nativeBlockNum, parityBlockNum, chunkSize);
	printf("Decoding file completed: %fms\n", stepTime);
	totalComputationTime += stepTime;

	// record event
	hipEventRecord(stepStart);
	hipMemcpy(dataBuf, dataBuf_d, dataSize, hipMemcpyDeviceToHost);
	// record event and synchronize
	hipEventRecord(stepStop);
	hipEventSynchronize(stepStop);
	// get event elapsed time
	hipEventElapsedTime(&stepTime, stepStart, stepStop);
	printf("copy data from GPU to CPU: %fms\n", stepTime);
	totalCommunicationTime += stepTime;

	hipFree(decodingMatrix_d);
	hipFree(dataBuf_d);
	hipFree(codeBuf_d);

	// record event and synchronize
	hipEventRecord(totalStop);
	hipEventSynchronize(totalStop);
	// get event elapsed time
	hipEventElapsedTime(&totalTime, totalStart, totalStop);
	printf("Total computation time: %fms\n", totalComputationTime);
	printf("Total communication time: %fms\n", totalCommunicationTime);
	printf("Total GPU decoding time: %fms\n", totalTime);
}

static void* GPU_thread_func(void * args)
{
	ThreadDataType* thread_data = (ThreadDataType *) args;
	hipSetDevice(thread_data->id);
	struct timespec start, end;
	pthread_barrier_wait(&barrier);
	clock_gettime(CLOCK_REALTIME, &start);
	pthread_barrier_wait(&barrier);
	decode(thread_data->dataBuf, 
			thread_data->codeBuf, 
			thread_data->decodingMatrix, 
			thread_data->id, 
			thread_data->nativeBlockNum, 
			thread_data->parityBlockNum, 
			thread_data->chunkSize);
	pthread_barrier_wait(&barrier);
	clock_gettime(CLOCK_REALTIME, &end);
	if (thread_data->id == 0)
	{
		double totalTime = (double) (end.tv_sec - start.tv_sec) * 1000
				+ (double) (end.tv_nsec - start.tv_nsec) / (double) 1000000L;
		printf("Total GPU decoding time using multiple devices: %fms\n", totalTime);
	}
	return NULL;
}

extern "C"
void decode_file(char *inFile, char *confFile, char *outFile)
{
	int chunkSize = 1;
	int totalSize;
	int parityBlockNum;
	int nativeBlockNum;

	uint8_t *dataBuf;		//host
	uint8_t *codeBuf;		//host

	int dataSize;
	int codeSize;

	FILE *fp_meta;
	FILE *fp_in;
	FILE *fp_out;

	int totalMatrixSize;
	int matrixSize;
	uint8_t *totalEncodingMatrix;	//host
	uint8_t *encodingMatrix;	//host
	char metadata_file_name[strlen(inFile) + 15];
	sprintf(metadata_file_name, "%s.METADATA", inFile);
	if((fp_meta = fopen(metadata_file_name, "rb")) == NULL)
	{
		printf("Cannot open metadata file!\n");
		exit(0);
	}
	fscanf(fp_meta, "%d", &totalSize);
	fscanf(fp_meta, "%d %d", &parityBlockNum, &nativeBlockNum);

	chunkSize = (totalSize / nativeBlockNum) + (totalSize%nativeBlockNum != 0); 
//	chunkSize = (int) (ceil((double) totalSize / nativeBlockNum)); 
#ifdef DEBUG
printf("chunk size: %d\n", chunkSize);
#endif

	totalMatrixSize = nativeBlockNum * (nativeBlockNum + parityBlockNum);
	totalEncodingMatrix = (uint8_t*) malloc(totalMatrixSize);
	matrixSize = nativeBlockNum * nativeBlockNum;
	encodingMatrix = (uint8_t*) malloc(matrixSize);
	for (int i = 0; i < totalMatrixSize; ++i)
	{
//		fscanf(fp_meta, "%d", &totalEncodingMatrix[i]);
		int j;
		fscanf(fp_meta, "%d", &j);
		totalEncodingMatrix[i] = (uint8_t) j; 
	}
	fclose(fp_meta);

	dataSize = nativeBlockNum * chunkSize * sizeof(uint8_t);
	codeSize = nativeBlockNum * chunkSize * sizeof(uint8_t);
	dataBuf = (uint8_t*) malloc(dataSize);
	memset(dataBuf, 0, dataSize);
	codeBuf = (uint8_t*) malloc(codeSize);
	memset(codeBuf, 0, codeSize);

	FILE *fp_conf;
	char input_file_name[strlen(inFile) + 20];
	int index;
	if((fp_conf = fopen(confFile, "r")) == NULL)
	{
		printf("Cannot open configuration file!\n");
		exit(0);
	}

	for(int i = 0; i < nativeBlockNum; i++)
	{
		fscanf(fp_conf, "%s", input_file_name);
		index = atoi(input_file_name + 1);

		copy_matrix(totalEncodingMatrix, encodingMatrix, index, i, nativeBlockNum);

		if((fp_in = fopen(input_file_name, "rb")) == NULL)
		{
			printf("Cannot open input file %s!\n", input_file_name);
			exit(0);
		}
		fseek(fp_in, 0L, SEEK_SET);
		// this part can be process in parallel with computing inversed matrix
		fread(codeBuf + i * chunkSize, sizeof(uint8_t), chunkSize, fp_in);
		fclose(fp_in);
	}
	fclose(fp_conf);
	
	uint8_t *decodingMatrix;
	decodingMatrix = (uint8_t*) malloc(matrixSize);
    CPU_invert_matrix(encodingMatrix, decodingMatrix, nativeBlockNum);

	int GPU_num;
	hipGetDeviceCount(&GPU_num);
	void* threads = malloc(GPU_num * sizeof(pthread_t));
	ThreadDataType* thread_data = (ThreadDataType *) malloc(GPU_num * sizeof(ThreadDataType));
	uint8_t *dataBufPerDevice[GPU_num];
	uint8_t *codeBufPerDevice[GPU_num];
	pthread_barrier_init(&barrier, NULL, GPU_num);
	int maxChunkSizePerDevice = (chunkSize / GPU_num) + (chunkSize % GPU_num != 0);
//	struct timespec start, end;
//	clock_gettime(CLOCK_REALTIME, &start);
	for (int i = 0; i < GPU_num; ++i)
	{
		thread_data[i].id = i;
		thread_data[i].nativeBlockNum = nativeBlockNum;
		thread_data[i].parityBlockNum = parityBlockNum;
		int deviceChunkSize = min(chunkSize - i * maxChunkSizePerDevice, maxChunkSizePerDevice);
		thread_data[i].chunkSize = deviceChunkSize;
		int deviceDataSize = nativeBlockNum * deviceChunkSize * sizeof(uint8_t);
		int deviceCodeSize = nativeBlockNum * deviceChunkSize * sizeof(uint8_t);
//		dataBufPerDevice[i] = (uint8_t*) malloc(deviceDataSize);
		hipHostMalloc((void **)&dataBufPerDevice[i], deviceDataSize);
//		codeBufPerDevice[i] = (uint8_t*) malloc(deviceCodeSize);
		hipHostMalloc((void **)&codeBufPerDevice[i], deviceCodeSize);
		for (int j = 0; j < nativeBlockNum; ++j)
		{
//			memcpy(codeBufPerDevice[i] + j * deviceChunkSize, 
//							codeBuf + j * chunkSize + i * deviceChunkSize,
//							deviceChunkSize);
			hipMemcpy(codeBufPerDevice[i] + j * deviceChunkSize, 
							codeBuf + j * chunkSize + i * deviceChunkSize,
							deviceChunkSize,
							hipMemcpyHostToHost);
		}
		thread_data[i].dataBuf = dataBufPerDevice[i];
		thread_data[i].codeBuf = codeBufPerDevice[i];
		thread_data[i].decodingMatrix = decodingMatrix;
		pthread_create(&((pthread_t*) threads)[i], NULL, GPU_thread_func, (void *) &thread_data[i]);
	}
	for (int i = 0; i < GPU_num; ++i)
	{
		pthread_join(((pthread_t*) threads)[i], NULL);
	}
//	clock_gettime(CLOCK_REALTIME, &end);
//	double totalTime = (double) (end.tv_sec - start.tv_sec) * 1000
//			+ (double) (end.tv_nsec - start.tv_nsec) / (double) 1000000L;
//	printf("Total GPU decoding time using multiple devices: %fms\n", totalTime);
	for (int i = 0; i < GPU_num; ++i)
	{
		int deviceChunkSize = min(chunkSize - i * maxChunkSizePerDevice, maxChunkSizePerDevice);
		for (int j = 0; j < nativeBlockNum; ++j)
		{
//			memcpy(dataBuf + j * chunkSize + i * deviceChunkSize,
//							dataBufPerDevice[i] + j * deviceChunkSize,
//							deviceChunkSize);
			hipMemcpy(dataBuf + j * chunkSize + i * deviceChunkSize,
							dataBufPerDevice[i] + j * deviceChunkSize,
							deviceChunkSize,
							hipMemcpyHostToHost);
		}
//		free(dataBufPerDevice[i]);
		hipHostFree(dataBufPerDevice[i]);
//		free(codeBufPerDevice[i]);
		hipHostFree(codeBufPerDevice[i]);
	}
	pthread_barrier_destroy(&barrier);
	hipDeviceReset();

	if(outFile == NULL)
	{
		if((fp_out = fopen(inFile, "wb")) == NULL)
		{
			printf("Cannot open output file %s!\n", inFile);
			exit(0);
		}
	}
	else
	{
		if((fp_out = fopen(outFile, "wb")) == NULL)
		{
			printf("Cannot open output file %s!\n", outFile);
			exit(0);
		}
	}
	fwrite(dataBuf, sizeof(uint8_t), totalSize, fp_out);
	fclose(fp_out);

	free(dataBuf);
	free(codeBuf);
}
