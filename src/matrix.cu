#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  matrix.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/21/2012 07:38:17 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Shuai YUAN (), 
 *        Company:  
 *
 * =====================================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include "matrix.h"
//#include "galoisfield.h"

//#define SQUARE_BLOCK_SIZE 16    // MAX 
//#define SINGLE_BLOCK_SIZE 512   // MAX 
//
//#define DISPLAY_SETTINGS false
//#define DISPLAY false
//
////#define IDC2D(i,j,ld) (((j)*(ld))+(i))
//#define IDC2D(i,j,ld) (((i)*(ld))+(j))
//
//#define TILE_WIDTH 4
//#define TILE_DEPTH 4
//
//#define W 8
//#define NW (1 << W) /* In other words, NW equals 2 to the w-th power */

__shared__ uint8_t gflog[256];
__shared__ uint8_t gfexp[256];

__host__ __device__ int setup_tables(int w)
{
	unsigned int b;
   	unsigned int log;
	unsigned int x_to_w;
	unsigned int prim_poly;
//	unsigned int r;
//	unsigned int x;
//	unsigned int y;

	unsigned int prim_poly_4 = 023;
	unsigned int prim_poly_8 = 0435;
	//uint8_t prim_poly_8 = 285;
	unsigned int prim_poly_16 = 0210013;
	switch(w) 
	{
		case 4: prim_poly = prim_poly_4; break;
		case 8: prim_poly = prim_poly_8; break;
		case 16: prim_poly = prim_poly_16; break;
		default: return -1;
	}
	x_to_w = 1 << w;
	b = 1;
//	r = 0;
	for (log = 0; log < x_to_w-1; log++) 
	{
		/*
		r = 0;
		x = 1;
		y = log;
		while(y)
		{
			printf("y=%d\n",y);
			if(y & 1)
			{
				r = r ^ b;
			}
			y = y >> 1;
			x = x << 1;
			if (x & x_to_w) x = x ^ prim_poly;
		}
			printf("r=%d\n",r);
			printf("log=%d\n",log);
		*/
		if(b>x_to_w) break;
		gflog[b] = (uint8_t) log;
		gfexp[log] = (uint8_t) b;
		b = b << 1;
		if (b & x_to_w) 
		{
			b = b ^ prim_poly;
		}
	}
	return 0;
}

__host__ __device__ uint8_t gf_add(uint8_t a, uint8_t b)
{
	return a^b;
}

__host__ __device__ uint8_t gf_sub(uint8_t a, uint8_t b)
{
	return gf_add(a, b);
}

__host__ __device__ uint8_t gf_mul(uint8_t a, uint8_t b)
{
	int sum_log;
	if (a == 0 || b == 0)
	{
		return 0;
	}
//	sum_log = (gflog[a] + gflog[b]) % (NW-1);
	sum_log = gflog[a] + gflog[b];
	if (sum_log >= NW-1)
	{	
		sum_log -= NW-1;
	}
	return gfexp[sum_log];
}

__host__ __device__ uint8_t gf_mul(uint8_t a, uint8_t b, uint8_t *gflog, uint8_t *gfexp)
{
	int sum_log;
	if (a == 0 || b == 0)
	{
		return 0;
	}
//	sum_log = (gflog[a] + gflog[b]) % (NW-1);
	sum_log = gflog[a] + gflog[b];
	if (sum_log >= NW-1)
	{	
		sum_log -= NW-1;
	}
	return gfexp[sum_log];
}

__host__ __device__ uint8_t gf_mul_bit(uint8_t a, uint8_t b)
{
	uint8_t sum_log;
	while(b)
	{
		if(b & 1)
		{
			sum_log ^= a;
		}
		a = (a << 1) ^ (a & 0x80? 0x1d: 0);
		b >>= 1;
	}
	return sum_log;
}

__host__ __device__ uint8_t gf_mul_bit(uint8_t a, uint8_t b, uint8_t *gflog, uint8_t *gfexp)
{
	uint8_t sum_log;
	while(b)
	{
		if(b & 1)
		{
			sum_log ^= a;
		}
		a = (a << 1) ^ (a & 0x80? 0x1d: 0);
		b >>= 1;
	}
	return sum_log;
}

__host__ __device__ uint8_t gf_div(uint8_t a, uint8_t b)
{
	int diff_log;
	if (a == 0)
	{	
		return 0;
	}
	/* Can’t divide by 0 */
	if (b == 0)
	{
		return -1;
	}
//	diff_log = (gflog[a] - gflog[b]) % (NW-1);
	diff_log = gflog[a] - gflog[b];
	if (diff_log < 0)
	{	
		diff_log += NW-1;
	}
	return gfexp[diff_log];
}

__host__ __device__ uint8_t gf_div(uint8_t a, uint8_t b, uint8_t *gflog, uint8_t *gfexp)
{
	int diff_log;
	if (a == 0)
	{	
		return 0;
	}
	/* Can’t divide by 0 */
	if (b == 0)
	{
		return -1;
	}
//	diff_log = (gflog[a] - gflog[b]) % (NW-1);
	diff_log = gflog[a] - gflog[b];
	if (diff_log < 0)
	{	
		diff_log += NW-1;
	}
	return gfexp[diff_log];
}

__host__ __device__ uint8_t gf_pow(uint8_t a, uint8_t power)
{
	int pow_log = (gflog[a] * power) % (NW-1);
	return gfexp[pow_log];
}

__host__ __device__ uint8_t gf_pow(uint8_t a, uint8_t power, uint8_t *gflog, uint8_t *gfexp)
{
	int pow_log = (gflog[a] * power) % (NW-1);
	return gfexp[pow_log];
}

// C=AB
// A: nxp
// B: pxm
// C: nxm
__device__ void matrix_mul(unsigned char *A, unsigned char *B, unsigned char *C, int n, int p, int m)
{
	__shared__ int rowVector[TILE_WIDTH_ROW][TILE_DEPTH];
	__shared__ int colVector[TILE_DEPTH][TILE_WIDTH_COL];
	__shared__ int product[TILE_WIDTH_ROW][TILE_WIDTH_COL];

	int bx = blockIdx.x;
   	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row;
	int col;
	int px;
	int py;	

	setup_tables(8);
	__syncthreads();

	for(py=ty; py<TILE_WIDTH_ROW; py+=blockDim.y)
	{
		for(px=tx; px<TILE_WIDTH_COL; px+=blockDim.x)
		{
			row = by*TILE_WIDTH_ROW+py;
			col = bx*TILE_WIDTH_COL+px;
			product[py][px] = 0;
			__syncthreads();
		
			for(int i=0; i<(int)(ceil((float)p/TILE_DEPTH)); i++)
			{
				for(int j=tx; j<TILE_DEPTH; j+=blockDim.x)
				{
					rowVector[py][j] = A[row*p+i*TILE_DEPTH+j];
				}
				for(int j=ty; j<TILE_DEPTH; j+=blockDim.y)
				{		
					colVector[j][px] = B[col+(i*TILE_DEPTH+j)*m];
				}
				__syncthreads();
		
				for(int j=0; j<TILE_DEPTH; j++)
				{
					product[py][px] ^= gf_mul(rowVector[py][j], colVector[j][px]);
//					dist[py][px] = gf_add(dist[py][px], gf_mul(rowVector[py][j], colVector[j][px]));
				}
				__syncthreads();
			}
			C[row*m+col] = product[py][px];
		}
	}
}


//// C=AB
//// A: nxp
//// B: pxm
//// C: nxm
//__device__ void matrix_mul(uint8_t *A, uint8_t *B, uint8_t *C, int n, int p, int m)
//{
//	__shared__ int rowVector[TILE_WIDTH][TILE_DEPTH];
//	__shared__ int colVector[TILE_DEPTH][TILE_WIDTH];
//	__shared__ int product[TILE_WIDTH][TILE_WIDTH];
//
//	int bx = blockIdx.x;
//   	int by = blockIdx.y;
//	int tx = threadIdx.x;
//	int ty = threadIdx.y;
//	int row;
//	int col;
//	int px;
//	int py;	
//
//	setup_tables(8);
//	__syncthreads();
//
//	for(py=ty; py<TILE_WIDTH; py+=blockDim.y)
//	{
//		for(px=tx; px<TILE_WIDTH; px+=blockDim.x)
//		{
//			row = by*TILE_WIDTH+py;
//			col = bx*TILE_WIDTH+px;
//			product[py][px] = 0;
//			__syncthreads();
//		
//			for(int i=0; i<(int)(ceil((float)p/TILE_DEPTH)); i++)
//			{
//				for(int j=tx; j<TILE_DEPTH; j+=blockDim.x)
//				{
//					rowVector[py][j] = A[row*p+i*TILE_DEPTH+j];
//				}
//				for(int j=ty; j<TILE_DEPTH; j+=blockDim.y)
//				{		
//					colVector[j][px] = B[col+(i*TILE_DEPTH+j)*m];
//				}
//				__syncthreads();
//		
//				for(int j=0; j<TILE_DEPTH; j++)
//				{
//					product[py][px] ^= gf_mul(rowVector[py][j], colVector[j][px]);
////					dist[py][px] = gf_add(dist[py][px], gf_mul(rowVector[py][j], colVector[j][px]));
//				}
//				__syncthreads();
//			}
//			C[row*m+col] = product[py][px];
//		}
//	}
//	/*
//	int i;
//	int j;
//	int k;
//	setup_tables(8);
//	for(i=0; i<n; i++)
//	{
//		for(j=0; j<m; j++)
//		{
//			for(k=0; k<p; k++)
//			{
//				C[i*m+j] = gf_add( C[i*m+j], gf_mul( A[i*p+k], B[k*m+j] ) );
//			}
//		}
//	}
//	*/
//}

/******************************************************************************
                           AWAKE THE GPU CARD KERNEL

                                    works
*******************************************************************************/
__global__ void initKernel(){}


/******************************************************************************
                        ROUND UP - AIDING FUNCTION

*******************************************************************************/
int roundUp ( int n, int d )
{
	return n/d + (n%d != 0);
}

/******************************************************************************
                        MINIMUM - AIDING FUNCTION

*******************************************************************************/
int minimo(     int a,
                int b )
{
    return ( a < b )? a : b;

}

/******************************************************************************
                   SWITCH ROWS IF NECESSARY - KERNEL 

            Needs only a "row" of threads, block can be linear!

                                works                                    
*******************************************************************************/
__global__ void switch_rows(uint8_t *matrix, uint8_t *result, int rowSrc, int rowDes, int size)
{
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    uint8_t oldMatrixItem;
    uint8_t oldResultItem;

    if( col < size )
    {
        oldMatrixItem = matrix[ IDC2D(rowSrc, col, size) ];
        matrix[ IDC2D(rowSrc, col, size) ] = matrix[ IDC2D(rowDes, col, size) ];
        matrix[ IDC2D(rowDes, col, size) ] = oldMatrixItem; 

        oldResultItem = result[ IDC2D(rowSrc, col, size) ];
        result[ IDC2D(rowSrc, col, size) ] = result[ IDC2D(rowDes, col, size) ];
        result[ IDC2D(rowDes, col, size) ] = oldResultItem; 
    }
} 
__global__ void switch_columns(uint8_t *matrix, uint8_t *result, int colSrc, int colDes, int size)
{
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    uint8_t oldMatrixItem;
    uint8_t oldResultItem;

    if( row < size )
    {
        oldMatrixItem = matrix[ IDC2D(row, colSrc, size) ];
        matrix[ IDC2D(row, colSrc, size) ] = matrix[ IDC2D(row, colDes, size) ];
        matrix[ IDC2D(row, colDes, size) ] = oldMatrixItem; 

        oldResultItem = result[ IDC2D(row, colSrc, size) ];
        result[ IDC2D(row, colSrc, size) ] = result[ IDC2D(row, colDes, size) ];
        result[ IDC2D(row, colSrc, size) ] = oldResultItem; 
    }
} 
/******************************************************************************
                    NORMALIZE THE PIVOT ROW KERNEL

            Needs only a "column" of threads, block can be linear!

                                works
*******************************************************************************/
__global__ void normalize_pivot_row(uint8_t *matrix, uint8_t *result, int row, int size)
{
    int ty = threadIdx.y;
	int col = blockDim.y*blockIdx.y + ty;

    __shared__ uint8_t pivotValue;

	setup_tables(8);
	__syncthreads();

    if( col < size )
    {
    	// let the first thread of loads the pivotValue
        if ( ty == 0 )
		{
            pivotValue = matrix[ IDC2D(row, row, size) ];
		}
        __syncthreads();
	// Normalize the pivot row!
	// Every thread divides the element of its position with the pivotValue
        matrix[ IDC2D(row, col, size)] = gf_div(matrix[ IDC2D(row, col, size) ], pivotValue);
        result[ IDC2D(row, col, size)] = gf_div(result[ IDC2D(row, col, size) ], pivotValue);
    }
}
__global__ void normalize_pivot_col(uint8_t *matrix, uint8_t *result, int col, int size)
{
    int ty = threadIdx.y;
	int row = blockDim.y*blockIdx.y + ty;

    __shared__ uint8_t pivotValue;

	setup_tables(8);
	__syncthreads();

    if( col < size )
    {
    	// let the first thread of loads the pivotValue
        if ( ty == 0 )
		{
            pivotValue = matrix[ IDC2D(col, col, size) ];
		}
        __syncthreads();
	// Normalize the pivot row!
	// Every thread divides the element of its position with the pivotValue
        matrix[ IDC2D(row, col, size)] = gf_div(matrix[ IDC2D(row, col, size) ], pivotValue);
        result[ IDC2D(row, col, size)] = gf_div(result[ IDC2D(row, col, size) ], pivotValue);
    }
}

/******************************************************************************
         MODIFIED GAUSSIAN ELIMINATION (MGE) KERNEL - LINEAR VERSION

        As the name implies, its grid is formed out of linear blocks

*******************************************************************************/

__global__ void eliminate_by_row(uint8_t *matrix, uint8_t *result, int pivotIndex, int size)
{
    int ty = threadIdx.y;

	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockIdx.x;

    __shared__ uint8_t pivotCol[ SINGLE_BLOCK_SIZE ];

    __shared__ uint8_t matrixPivotValue;
    __shared__ uint8_t resultPivotValue;
    __shared__ uint8_t matrixCol[ SINGLE_BLOCK_SIZE ];
    __shared__ uint8_t resultCol[ SINGLE_BLOCK_SIZE];

	setup_tables(8);
	__syncthreads();

    if ( row < size )
    {
        if ( ty == 0 )
        {
            matrixPivotValue = matrix[ IDC2D(pivotIndex, col, size) ];
            resultPivotValue = result[ IDC2D(pivotIndex, col, size) ];
        }
        pivotCol[ty] = matrix[ IDC2D(row, pivotIndex, size) ];
        
        matrixCol[ty] = matrix[ IDC2D(row, col, size) ]; 
        resultCol[ty] = result[ IDC2D(row, col, size) ]; 
        __syncthreads();

		// substraction in GF
		// make the pivotCol become reduced echelon form
        if ( row != pivotIndex )
        {
			matrix[ IDC2D(row, col, size) ] = matrixCol[ty] ^ gf_mul(pivotCol[ty], matrixPivotValue);
			result[ IDC2D(row, col, size) ] = resultCol[ty] ^ gf_mul(pivotCol[ty], resultPivotValue);
        }
    }
}

__global__ void eliminate_by_col(uint8_t *matrix, uint8_t *result, int pivotIndex, int size)
{
    int ty = threadIdx.y;

	int row = blockIdx.x;
	int col = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ uint8_t pivotRow[ SINGLE_BLOCK_SIZE ];

    __shared__ uint8_t matrixPivotValue;
    __shared__ uint8_t resultPivotValue;
    __shared__ uint8_t matrixCol[ SINGLE_BLOCK_SIZE ];
    __shared__ uint8_t resultCol[ SINGLE_BLOCK_SIZE];

	setup_tables(8);
	__syncthreads();

    if ( row < size )
    {
        if ( ty == 0 )
        {
            matrixPivotValue = matrix[ IDC2D(row, pivotIndex, size) ];
            resultPivotValue = result[ IDC2D(row, pivotIndex, size) ];
        }
        pivotRow[ty] = matrix[ IDC2D(pivotIndex, col, size) ];
        
        matrixCol[ty] = matrix[ IDC2D(row, col, size) ]; 
        resultCol[ty] = result[ IDC2D(row, col, size) ]; 
        __syncthreads();

		// substraction in GF
		// make the pivotCol become reduced echelon form
        if ( col != pivotIndex )
        {
			matrix[ IDC2D(row, col, size) ] = matrixCol[ty] ^ gf_mul(pivotRow[ty], matrixPivotValue);
			result[ IDC2D(row, col, size) ] = resultCol[ty] ^ gf_mul(pivotRow[ty], resultPivotValue);
        }
    }
}




/******************************************************************************
                        CREATE RESULT KERNEL 

    Needs a complete grid of threads for ALL of the matrix positions.
    
    * SQUARE_BLOCK_SIZE and squareGrid used!

*******************************************************************************/

__global__ void get_identity_matrix(uint8_t *result, int size)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

    if ( row == col )
	{
        result[ IDC2D(row, col, size) ] = 1;
	}
    else
	{
        result[ IDC2D(row, col, size) ] = 0;
	}
}

/******************************************************************************
              GET THE MAXIMUM INDEX OF THE COLUMN - HOST CODE

                                    works
*******************************************************************************/
int get_pivot_index(uint8_t *row, int index, int size)
{
    int pivotIndex = -1;
    int i = index;
    while( pivotIndex == -1 && i < size )
    {
        pivotIndex = (row[i] > 0)? i: -1;        
        i++;
    }
    return pivotIndex;
}


/******************************************************************************
                MODIFIED GAUSSIAN ELIMINATION (MGE) - HOST CODE

*******************************************************************************/

extern "C"
void invert_matrix(uint8_t *matrix_dev, uint8_t *result_dev, int size)
{
	int row;
	int pivotIndex;
    uint8_t currentRow[size];
    int currentRowSize = size*sizeof(uint8_t);

/*
// setup for get_identity_matrix kernel (cr)
    dim3 crGrid( ceil( size, SQUARE_BLOCK_SIZE ), ceil( size, SQUARE_BLOCK_SIZE ) );    // GRID
    dim3 crBlock( minimo( size, SQUARE_BLOCK_SIZE ), minimo( size, SQUARE_BLOCK_SIZE ) );     // BLOCK

// setup for normalize_pivot_col kernel (npr) GRID
    dim3 nprGrid( 1, ceil( size, SINGLE_BLOCK_SIZE) );
 
// setup for linearMGE kernel (lmge) GRID
    dim3 lmgeGrid( size, ceil( size, SINGLE_BLOCK_SIZE));

// setup for normalize_pivot_col kernel (npr) AND linearMGE kernel BLOCK
    dim3 linearBlock( 1, minimo(size, SINGLE_BLOCK_SIZE)); 

// Shows the setup if DISPLAY_SETTINGS macro is TRUE
    if ( DISPLAY_SETTINGS )
    {
        printf( "\nKernels Setup:\n" );

        printf( "\t> crGrid( %d, %d )\n", crGrid.x, crGrid.y );
        printf( "\t> crBlock( %d, %d )\n", crBlock.x, crBlock.y );    

        printf( "\t> nprGrid( %d, %d )\n", nprGrid.x, nprGrid.y );
        printf( "\t> nprBlock( %d, %d )\n", linearBlock.x, linearBlock.y );

        printf( "\t> lmgeGrid( %d, %d )\n", lmgeGrid.x, lmgeGrid.y );
        printf( "\t> lmgeBlock( %d, %d )\n\n", linearBlock.x, linearBlock.y );
    }
*/
    dim3 gimGrid( (int)(ceil( (float)size / SQUARE_BLOCK_SIZE)), (int)(ceil( (float)size / SQUARE_BLOCK_SIZE)) );
    dim3 gimBlock( min(size, SQUARE_BLOCK_SIZE), min(size, SQUARE_BLOCK_SIZE) );
    get_identity_matrix<<< gimGrid, gimBlock >>>(result_dev, size);
//	hipDeviceSynchronize();
	
	for( row = 0; row < size; row++ )
    {
		// check whether the leading coefficient of the current row is in the 'index'th column
		int index = row;
        hipMemcpy(currentRow, matrix_dev+row*size, currentRowSize, hipMemcpyDeviceToHost);
        pivotIndex = get_pivot_index(currentRow, index, size);
        if( pivotIndex != row )
		{
			dim3 scGrid(1, (int)(ceil( (float)size / SINGLE_BLOCK_SIZE )));
			dim3 scBlock(1, min(size, SINGLE_BLOCK_SIZE)); 
            switch_columns<<< scGrid, scBlock >>>(matrix_dev, result_dev, index, pivotIndex, size);
		}
		hipDeviceSynchronize();
//		hipDeviceSynchronize();

		dim3 nprGrid(1, (int)(ceil( (float)size / SINGLE_BLOCK_SIZE )));
		dim3 nprBlock(1, min(size, SINGLE_BLOCK_SIZE)); 
    	// Normalize the pivot row
        normalize_pivot_row<<< nprGrid, nprBlock >>>(matrix_dev, result_dev, index, size);
//    	// Normalize the pivot column
//        normalize_pivot_col<<< nprGrid, linearBlock >>>(matrix_dev, result_dev, index, size);
		hipDeviceSynchronize();
//		hipDeviceSynchronize();

		dim3 ebrGrid(size, (int)(ceil( (float)size / SINGLE_BLOCK_SIZE )));
		dim3 ebrBlock(1, min(size, SINGLE_BLOCK_SIZE)); 
        eliminate_by_row<<< ebrGrid, ebrBlock >>>(matrix_dev, result_dev, row, size);
		hipDeviceSynchronize();
//		hipDeviceSynchronize();
    }
//    hipDeviceSynchronize();

}

__global__ void gen_encoding_matrix(uint8_t *encodingMatrix, int row, int col)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	setup_tables(8);
	__syncthreads();
	encodingMatrix[i*col + j] = gf_pow(j+1, i);
}

__global__ void encode_chunk(unsigned char *dataChunk, unsigned char *parityCoeff, unsigned char *codeChunk, int nativeBlockNum, int parityBlockNum, int chunkSize)
{
	matrix_mul(parityCoeff, dataChunk, codeChunk, parityBlockNum, nativeBlockNum, chunkSize);
}

__global__ void decode_chunk(unsigned char *dataChunk, unsigned char *parityCoeff, unsigned char *codeChunk, int nativeBlockNum, int parityBlockNum, int chunkSize)
{
	matrix_mul(parityCoeff, codeChunk, dataChunk, nativeBlockNum, nativeBlockNum, chunkSize);
}




