#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  encode.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/05/2012 10:42:32 PM
 *       Revision:  none
 *       Compiler:  nvcc/ggc
 *
 *         Author:  Shuai YUAN (), 
 *        Company:  
 *
 * =====================================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include "matrix.h"

//#define W 8
//#define NW (1 << W) /* In other words, NW equals 2 to the w-th power */

//#define TILE_WIDTH 2
//#define TILE_DEPTH 2

//#define BUFFER_SIZE 256

//#define DEBUG 

//__global__ void encode_chunk(unsigned char *dataChunk, unsigned char *parityCoeff, unsigned char *codeChunk, int nativeBlockNum, int parityBlockNum, int chunkSize)
//{
//	matrix_mul(parityCoeff, dataChunk, codeChunk, parityBlockNum, nativeBlockNum, chunkSize);
//}
//
//__global__ void gen_encoding_matrix(uint8_t *encodingMatrix, int row, int col)
//{
//	int i = threadIdx.x;
//	int j = threadIdx.y;
//	setup_tables(8);
//	__syncthreads();
//	encodingMatrix[i*col + j] = gf_pow(j+1, i);
//}


void write_metadata(int totalSize, int parityBlockNum, int nativeBlockNum, uint8_t *encodingMatrix)
{
	FILE *fp;
	if( ( fp = fopen(".METADATA", "wb") ) == NULL )
	{
		printf("Can not open META file!\n");
		exit(0);
	}
	fprintf(fp, "%d\n", totalSize);
	fprintf(fp, "%d %d\n", parityBlockNum, nativeBlockNum);
	int i;
	int j;
	for(i=0; i<nativeBlockNum; i++)
	{
		for(j=0; j<nativeBlockNum; j++)
		{
			if(i == j)
			{
				fprintf(fp, "1 ");
			}
			else
			{
				fprintf(fp, "0 ");
			}
		}
		fprintf(fp, "\n");
	}
	for(i=0; i<parityBlockNum; i++)
	{
		for(j=0; j<nativeBlockNum; j++)
		{
			fprintf(fp, "%d ", encodingMatrix[i*nativeBlockNum+j]);
		}
		fprintf(fp, "\n");
	}
	fclose(fp);
}

extern "C"
void encode(uint8_t *dataBuf, uint8_t *codeBuf, int nativeBlockNum, int parityBlockNum, int chunkSize, int totalSize)
{
	uint8_t *dataBuf_d;		//device
	uint8_t *codeBuf_d;		//device
	int dataSize = nativeBlockNum*chunkSize*sizeof(uint8_t);
	int codeSize = parityBlockNum*chunkSize*sizeof(uint8_t);
	hipMalloc( (void **)&dataBuf_d, nativeBlockNum*chunkSize*sizeof(uint8_t) );
//	hipMemset(dataBuf_d, 0, dataSize);
	hipMalloc( (void **)&codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t) );
//	hipMemset(codeBuf_d, 0, codeSize);

	hipMemcpy(dataBuf_d, dataBuf, nativeBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyHostToDevice);

	uint8_t *encodingMatrix;	//host
	uint8_t *encodingMatrix_d;	//device
	encodingMatrix = (uint8_t*) malloc( parityBlockNum*nativeBlockNum*sizeof(uint8_t) );
	hipMalloc( (void **)&encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t) );

	dim3 blk(parityBlockNum, nativeBlockNum);
	gen_encoding_matrix<<<1, blk>>>(encodingMatrix_d, parityBlockNum, nativeBlockNum);
//	hipDeviceSynchronize();

	hipMemcpy(encodingMatrix, encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t), hipMemcpyDeviceToHost);
	write_metadata(totalSize, parityBlockNum, nativeBlockNum, encodingMatrix);

//	int gridDimX = (int)(ceil((float)chunkSize/TILE_WIDTH));
//	int gridDimY = (int)(ceil((float)parityBlockNum/TILE_WIDTH));
//	dim3 grid(gridDimX, gridDimY);
//	dim3 block(TILE_WIDTH, TILE_WIDTH);
	int gridDimX = (int)( ceil((float)chunkSize / TILE_WIDTH_COL) );
	int gridDimY = (int)( ceil((float)nativeBlockNum / TILE_WIDTH_ROW) );
	dim3 grid(gridDimX, gridDimY);
	dim3 block(TILE_WIDTH_ROW, TILE_WIDTH_COL);
	encode_chunk<<<grid, block>>>(dataBuf_d, encodingMatrix_d, codeBuf_d, nativeBlockNum, parityBlockNum, chunkSize);

	hipMemcpy(codeBuf, codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(encodingMatrix_d);
	hipFree(dataBuf_d);
	hipFree(codeBuf_d);

	free(encodingMatrix);

}

extern "C"
void encode_file(char *file, int nativeBlockNum, int parityBlockNum)
{
	int chunkSize = 1;
	int totalSize;

	FILE *fp_in;
	FILE *fp_out;
	if( ( fp_in = fopen(file,"rb") ) == NULL )
	{
		printf("Can not open source file!\n");
		exit(0);
	}

	fseek(fp_in, 0L, SEEK_END);
	//ftell() get the total size of the file
	totalSize = ftell(fp_in);
	chunkSize = (int) (ceil( (float)ftell(fp_in) / nativeBlockNum )); 

	uint8_t *dataBuf;		//host
	uint8_t *codeBuf;		//host
	int dataSize = nativeBlockNum*chunkSize*sizeof(uint8_t);
	int codeSize = parityBlockNum*chunkSize*sizeof(uint8_t);
	dataBuf = (uint8_t*) malloc( nativeBlockNum*chunkSize*sizeof(uint8_t) );
	memset(dataBuf, 0, dataSize);
	codeBuf = (uint8_t*) malloc( parityBlockNum*chunkSize*sizeof(uint8_t) );
	memset(codeBuf, 0, codeSize);
	
	int i;
	for(i=0; i<nativeBlockNum; i++)
	{
		if( fseek( fp_in, i*chunkSize, SEEK_SET ) == -1 )
		{
			printf("fseek error!\n");
			exit(0);
		}

		if( fread( dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_in ) == EOF )
		{
			printf("fread error!\n");
			exit(0);
		}
	}
	fclose(fp_in);
	
	encode(dataBuf, codeBuf, nativeBlockNum, parityBlockNum, chunkSize, totalSize);

	char output_file_name[20];
	for(i=0; i<nativeBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_", i);
		strcat(output_file_name, file);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open source file!\n");
			exit(0);
		}
		if( fwrite(dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}
	for(i=0; i<parityBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_", i+nativeBlockNum);
		strcat(output_file_name, file);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open source file!\n");
			exit(0);
		}
		if( fwrite(codeBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}

	free(dataBuf);
	free(codeBuf);
}
/*
int main(int argc, char *argv[])
{
	int nativeBlockNum = 4;
	int parityBlockNum = 2;
//	int chunkSize = sizeof(uint8_t);
	int chunkSize = 1;
	int totalSize;

	FILE *fp_in;
	FILE *fp_out;
	if( ( fp_in = fopen(argv[1],"rb") ) == NULL )
	{
		printf("Can not open source file!\n");
		exit(0);
	}

	fseek(fp_in, 0L, SEEK_END);
	//ftell() get the total size of the file
	totalSize = ftell(fp_in);
	chunkSize = (int) (ceil( (float)ftell(fp_in) / nativeBlockNum )); 

	uint8_t *dataBuf;		//host
	uint8_t *codeBuf;		//host
	uint8_t *dataBuf_d;		//device
	uint8_t *codeBuf_d;		//device
	int dataSize = nativeBlockNum*chunkSize*sizeof(uint8_t);
	int codeSize = parityBlockNum*chunkSize*sizeof(uint8_t);
	dataBuf = (uint8_t*) malloc( nativeBlockNum*chunkSize*sizeof(uint8_t) );
	memset(dataBuf, 0, dataSize);
	codeBuf = (uint8_t*) malloc( parityBlockNum*chunkSize*sizeof(uint8_t) );
	memset(codeBuf, 0, codeSize);
	hipMalloc( (void **)&dataBuf_d, nativeBlockNum*chunkSize*sizeof(uint8_t) );
	hipMemset(dataBuf_d, 0, dataSize);
	hipMalloc( (void **)&codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t) );
	hipMemset(codeBuf_d, 0, codeSize);

	int i;
	for(i=0; i<nativeBlockNum; i++)
	{
		if( fseek( fp_in, i*chunkSize, SEEK_SET) == -1 )
		{
			printf("fseek error!\n");
			exit(0);
		}

		if( fread( dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_in ) == EOF )
		{
			printf("fread error!\n");
			exit(0);
		}
	}
	fclose(fp_in);

	hipMemcpy(dataBuf_d, dataBuf, nativeBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyHostToDevice);

	// setup table for GF(2^8)
//	uint8_t *gflog;	//host
//	uint8_t *gfexp;	//host
//	uint8_t *gflog_d;	//device
//	uint8_t *gfexp_d;	//device
//	gflog = (uint8_t*) malloc( 512*sizeof(uint8_t) );
//	gfexp = (uint8_t*) malloc( 512*sizeof(uint8_t) );
//	hipMalloc( (void **)&gflog_d, 512*sizeof(uint8_t) );
//	hipMalloc( (void **)&gfexp_d, 512*sizeof(uint8_t) );
//	setup_tables(8);
//	hipDeviceSynchronize();
//	hipMemcpy(gflog_d, gflog, 512*sizeof(uint8_t), hipMemcpyDeviceToHost);

	uint8_t *encodingMatrix;	//host
	uint8_t *encodingMatrix_d;	//device
	encodingMatrix = (uint8_t*) malloc( parityBlockNum*nativeBlockNum*sizeof(uint8_t) );
	hipMalloc( (void **)&encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t) );
	dim3 blk(parityBlockNum, nativeBlockNum);
	gen_encoding_matrix<<<1, blk>>>(encodingMatrix_d, parityBlockNum, nativeBlockNum);
//	hipDeviceSynchronize();

	hipMemcpy(encodingMatrix, encodingMatrix_d, parityBlockNum*nativeBlockNum*sizeof(uint8_t), hipMemcpyDeviceToHost);
	write_metadata(totalSize, parityBlockNum, nativeBlockNum, encodingMatrix);

	int gridDimX = (int)(ceil((float)chunkSize/TILE_WIDTH));
	int gridDimY = (int)(ceil((float)parityBlockNum/TILE_WIDTH));
	dim3 grid(gridDimX, gridDimY);
	dim3 block(TILE_WIDTH, TILE_WIDTH);

	encode_chunk<<<grid, block>>>(dataBuf_d, encodingMatrix_d, codeBuf_d, nativeBlockNum, parityBlockNum, chunkSize);
	hipMemcpy(codeBuf, codeBuf_d, parityBlockNum*chunkSize*sizeof(uint8_t), hipMemcpyDeviceToHost);


#ifdef DEBUG
//	show_code_chunk(codeBuf, parityBlockNum, chunkSize);
#endif

	char output_file_name[20];

	for(i=0; i<nativeBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_", i);
		strcat(output_file_name, argv[1]);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open source file!\n");
			exit(0);
		}
		if( fwrite(dataBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}

	for(i=0; i<parityBlockNum; i++)
	{
		sprintf(output_file_name, "_%d_", i+nativeBlockNum);
		strcat(output_file_name, argv[1]);
		if( ( fp_out = fopen(output_file_name, "wb") ) == NULL )
		{
			printf("Can not open source file!\n");
			exit(0);
		}
		if( fwrite(codeBuf+i*chunkSize, sizeof(uint8_t), chunkSize, fp_out ) != sizeof(uint8_t)*chunkSize )
		{
			printf("fwrite error!\n");
			exit(0);
		}
		fclose(fp_out);
	}

	hipFree(encodingMatrix_d);
	hipFree(dataBuf_d);
	hipFree(codeBuf_d);

	free(encodingMatrix);
	free(dataBuf);
	free(codeBuf);

}
*/
