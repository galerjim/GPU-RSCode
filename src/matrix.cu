#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  matrix.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  12/21/2012 07:38:17 PM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Shuai YUAN (yszheda AT gmail.com), 
 *        Company:  
 *
 * =====================================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include "matrix.h"

__shared__ uint8_t gflog[256];
__shared__ uint8_t gfexp[256];

__host__ __device__ int setup_tables(int w)
{
	unsigned int b;
   	unsigned int log;
	unsigned int x_to_w;
	unsigned int prim_poly;
//	unsigned int r;
//	unsigned int x;
//	unsigned int y;

	unsigned int prim_poly_4 = 023;
	unsigned int prim_poly_8 = 0435;
	//uint8_t prim_poly_8 = 285;
	unsigned int prim_poly_16 = 0210013;
	switch(w) 
	{
		case 4: prim_poly = prim_poly_4; break;
		case 8: prim_poly = prim_poly_8; break;
		case 16: prim_poly = prim_poly_16; break;
		default: return -1;
	}
	x_to_w = 1 << w;
	b = 1;
//	r = 0;
	for (log = 0; log < x_to_w-1; log++) 
	{
		/*
		r = 0;
		x = 1;
		y = log;
		while(y)
		{
			printf("y=%d\n",y);
			if(y & 1)
			{
				r = r ^ b;
			}
			y = y >> 1;
			x = x << 1;
			if (x & x_to_w) x = x ^ prim_poly;
		}
			printf("r=%d\n",r);
			printf("log=%d\n",log);
		*/
		if(b>x_to_w) break;
		gflog[b] = (uint8_t) log;
		gfexp[log] = (uint8_t) b;
		b = b << 1;
		if (b & x_to_w) 
		{
			b = b ^ prim_poly;
		}
	}
	return 0;
}

__host__ __device__ uint8_t gf_add(uint8_t a, uint8_t b)
{
	return a^b;
}

__host__ __device__ uint8_t gf_sub(uint8_t a, uint8_t b)
{
	return gf_add(a, b);
}

__host__ __device__ uint8_t gf_mul(uint8_t a, uint8_t b)
{
	int sum_log;
	if (a == 0 || b == 0)
	{
		return 0;
	}
//	sum_log = (gflog[a] + gflog[b]) % (NW-1);
	sum_log = gflog[a] + gflog[b];
	if (sum_log >= NW-1)
	{	
		sum_log -= NW-1;
	}
	return gfexp[sum_log];
}

__host__ __device__ uint8_t gf_mul(uint8_t a, uint8_t b, uint8_t *gflog, uint8_t *gfexp)
{
	int sum_log;
	if (a == 0 || b == 0)
	{
		return 0;
	}
//	sum_log = (gflog[a] + gflog[b]) % (NW-1);
	sum_log = gflog[a] + gflog[b];
	if (sum_log >= NW-1)
	{	
		sum_log -= NW-1;
	}
	return gfexp[sum_log];
}

__host__ __device__ uint8_t gf_mul_bit(uint8_t a, uint8_t b)
{
	uint8_t sum_log;
	while(b)
	{
		if(b & 1)
		{
			sum_log ^= a;
		}
		a = (a << 1) ^ (a & 0x80? 0x1d: 0);
		b >>= 1;
	}
	return sum_log;
}

__host__ __device__ uint8_t gf_mul_bit(uint8_t a, uint8_t b, uint8_t *gflog, uint8_t *gfexp)
{
	uint8_t sum_log;
	while(b)
	{
		if(b & 1)
		{
			sum_log ^= a;
		}
		a = (a << 1) ^ (a & 0x80? 0x1d: 0);
		b >>= 1;
	}
	return sum_log;
}

__host__ __device__ uint8_t gf_div(uint8_t a, uint8_t b)
{
	int diff_log;
	if (a == 0)
	{	
		return 0;
	}
	/* Can’t divide by 0 */
	if (b == 0)
	{
		return -1;
	}
//	diff_log = (gflog[a] - gflog[b]) % (NW-1);
	diff_log = gflog[a] - gflog[b];
	if (diff_log < 0)
	{	
		diff_log += NW-1;
	}
	return gfexp[diff_log];
}

__host__ __device__ uint8_t gf_div(uint8_t a, uint8_t b, uint8_t *gflog, uint8_t *gfexp)
{
	int diff_log;
	if (a == 0)
	{	
		return 0;
	}
	/* Can’t divide by 0 */
	if (b == 0)
	{
		return -1;
	}
//	diff_log = (gflog[a] - gflog[b]) % (NW-1);
	diff_log = gflog[a] - gflog[b];
	if (diff_log < 0)
	{	
		diff_log += NW-1;
	}
	return gfexp[diff_log];
}

__host__ __device__ uint8_t gf_pow(uint8_t a, uint8_t power)
{
	int pow_log = (gflog[a] * power) % (NW-1);
	return gfexp[pow_log];
}

__host__ __device__ uint8_t gf_pow(uint8_t a, uint8_t power, uint8_t *gflog, uint8_t *gfexp)
{
	int pow_log = (gflog[a] * power) % (NW-1);
	return gfexp[pow_log];
}

// input matrix A and B, compute the product matrix C=AB
// A: nxp
// B: pxm
// C: nxm
__device__ void matrix_mul(unsigned char *A, unsigned char *B, unsigned char *C, int n, int p, int m)
{
	__shared__ int rowVector[TILE_WIDTH_ROW][TILE_DEPTH];
	__shared__ int colVector[TILE_DEPTH][TILE_WIDTH_COL];
	__shared__ int product[TILE_WIDTH_ROW][TILE_WIDTH_COL];

	int bx = blockIdx.x;
   	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row;
	int col;
	int px;
	int py;	

	setup_tables(8);
	__syncthreads();

for(bx=blockIdx.x; bx< (int)(ceil((float)m/gridDim.x)); bx+=gridDim.x )
{
	for(py=ty; py<TILE_WIDTH_ROW; py+=blockDim.y)
	{
		for(px=tx; px<TILE_WIDTH_COL; px+=blockDim.x)
		{
			row = by*TILE_WIDTH_ROW+py;
			col = bx*TILE_WIDTH_COL+px;
			product[py][px] = 0;
			__syncthreads();
		
if(row < n && col < m)
{
			for(int i=0; i<(int)(ceil((float)p/TILE_DEPTH)); i++)
			{
				int bound = min(p, TILE_DEPTH);
/*
				for(int j=tx; j<TILE_DEPTH; j+=blockDim.x)
				{
					rowVector[py][j] = A[row*p+i*TILE_DEPTH+j];
				}
				for(int j=ty; j<TILE_DEPTH; j+=blockDim.y)
				{		
					colVector[j][px] = B[col+(i*TILE_DEPTH+j)*m];
				}
				__syncthreads();
		
				for(int j=0; j<TILE_DEPTH; j++)
				{
					product[py][px] ^= gf_mul(rowVector[py][j], colVector[j][px]);
//					dist[py][px] = gf_add(dist[py][px], gf_mul(rowVector[py][j], colVector[j][px]));
				}
*/
				for(int j=tx; j<bound; j+=blockDim.x)
				{
					rowVector[py][j] = A[row*p+i*bound+j];
				}
				for(int j=ty; j<bound; j+=blockDim.y)
				{		
					colVector[j][px] = B[col+(i*bound+j)*m];
				}
				__syncthreads();
		
				for(int j=0; j<bound; j++)
				{
					product[py][px] ^= gf_mul(rowVector[py][j], colVector[j][px]);
//					dist[py][px] = gf_add(dist[py][px], gf_mul(rowVector[py][j], colVector[j][px]));
				}
				__syncthreads();
			}
			C[row*m+col] = product[py][px];
}
}
		}
	}
}


//// C=AB
//// A: nxp
//// B: pxm
//// C: nxm
//__device__ void matrix_mul(uint8_t *A, uint8_t *B, uint8_t *C, int n, int p, int m)
//{
//	__shared__ int rowVector[TILE_WIDTH][TILE_DEPTH];
//	__shared__ int colVector[TILE_DEPTH][TILE_WIDTH];
//	__shared__ int product[TILE_WIDTH][TILE_WIDTH];
//
//	int bx = blockIdx.x;
//   	int by = blockIdx.y;
//	int tx = threadIdx.x;
//	int ty = threadIdx.y;
//	int row;
//	int col;
//	int px;
//	int py;	
//
//	setup_tables(8);
//	__syncthreads();
//
//	for(py=ty; py<TILE_WIDTH; py+=blockDim.y)
//	{
//		for(px=tx; px<TILE_WIDTH; px+=blockDim.x)
//		{
//			row = by*TILE_WIDTH+py;
//			col = bx*TILE_WIDTH+px;
//			product[py][px] = 0;
//			__syncthreads();
//		
//			for(int i=0; i<(int)(ceil((float)p/TILE_DEPTH)); i++)
//			{
//				for(int j=tx; j<TILE_DEPTH; j+=blockDim.x)
//				{
//					rowVector[py][j] = A[row*p+i*TILE_DEPTH+j];
//				}
//				for(int j=ty; j<TILE_DEPTH; j+=blockDim.y)
//				{		
//					colVector[j][px] = B[col+(i*TILE_DEPTH+j)*m];
//				}
//				__syncthreads();
//		
//				for(int j=0; j<TILE_DEPTH; j++)
//				{
//					product[py][px] ^= gf_mul(rowVector[py][j], colVector[j][px]);
////					dist[py][px] = gf_add(dist[py][px], gf_mul(rowVector[py][j], colVector[j][px]));
//				}
//				__syncthreads();
//			}
//			C[row*m+col] = product[py][px];
//		}
//	}
//	/*
//	int i;
//	int j;
//	int k;
//	setup_tables(8);
//	for(i=0; i<n; i++)
//	{
//		for(j=0; j<m; j++)
//		{
//			for(k=0; k<p; k++)
//			{
//				C[i*m+j] = gf_add( C[i*m+j], gf_mul( A[i*p+k], B[k*m+j] ) );
//			}
//		}
//	}
//	*/
//}

// switch rows if the current row is not the pivot row
__global__ void switch_rows(uint8_t *matrix, uint8_t *result, int rowSrc, int rowDes, int size)
{
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    uint8_t oldMatrixItem;
    uint8_t oldResultItem;

    if( col < size )
    {
        oldMatrixItem = matrix[ IDC2D(rowSrc, col, size) ];
        matrix[ IDC2D(rowSrc, col, size) ] = matrix[ IDC2D(rowDes, col, size) ];
        matrix[ IDC2D(rowDes, col, size) ] = oldMatrixItem; 

        oldResultItem = result[ IDC2D(rowSrc, col, size) ];
        result[ IDC2D(rowSrc, col, size) ] = result[ IDC2D(rowDes, col, size) ];
        result[ IDC2D(rowDes, col, size) ] = oldResultItem; 
    }
} 
// switch columns if the current row is not the pivot row
__global__ void switch_columns(uint8_t *matrix, uint8_t *result, int colSrc, int colDes, int size)
{
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    uint8_t oldMatrixItem;
    uint8_t oldResultItem;

    if( row < size )
    {
        oldMatrixItem = matrix[ IDC2D(row, colSrc, size) ];
        matrix[ IDC2D(row, colSrc, size) ] = matrix[ IDC2D(row, colDes, size) ];
        matrix[ IDC2D(row, colDes, size) ] = oldMatrixItem; 

        oldResultItem = result[ IDC2D(row, colSrc, size) ];
        result[ IDC2D(row, colSrc, size) ] = result[ IDC2D(row, colDes, size) ];
        result[ IDC2D(row, colSrc, size) ] = oldResultItem; 
    }
} 

// normalize the row by the pivot value
__global__ void normalize_pivot_row(uint8_t *matrix, uint8_t *result, int row, int size)
{
    int ty = threadIdx.y;
	int col = blockDim.y*blockIdx.y + ty;

    __shared__ uint8_t pivotValue;

	setup_tables(8);
	__syncthreads();

    if( col < size )
    {
    	// let the first thread of loads the pivotValue
        if ( ty == 0 )
		{
            pivotValue = matrix[ IDC2D(row, row, size) ];
		}
        __syncthreads();
	// Normalize the pivot row!
	// Every thread divides the element of its position with the pivotValue
        matrix[ IDC2D(row, col, size)] = gf_div(matrix[ IDC2D(row, col, size) ], pivotValue);
        result[ IDC2D(row, col, size)] = gf_div(result[ IDC2D(row, col, size) ], pivotValue);
    }
}
// normalize the column by the pivot value
__global__ void normalize_pivot_col(uint8_t *matrix, uint8_t *result, int col, int size)
{
    int ty = threadIdx.y;
	int row = blockDim.y*blockIdx.y + ty;

    __shared__ uint8_t pivotValue;

	setup_tables(8);
	__syncthreads();

    if( col < size )
    {
    	// let the first thread of loads the pivotValue
        if ( ty == 0 )
		{
            pivotValue = matrix[ IDC2D(col, col, size) ];
		}
        __syncthreads();
	// Normalize the pivot row!
	// Every thread divides the element of its position with the pivotValue
        matrix[ IDC2D(row, col, size)] = gf_div(matrix[ IDC2D(row, col, size) ], pivotValue);
        result[ IDC2D(row, col, size)] = gf_div(result[ IDC2D(row, col, size) ], pivotValue);
    }
}

//eliminate by row to make the pivot column become reduced echelon form
__global__ void eliminate_by_row(uint8_t *matrix, uint8_t *result, int pivotIndex, int size)
{
    int ty = threadIdx.y;

	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockIdx.x;

    __shared__ uint8_t pivotCol[ SINGLE_BLOCK_SIZE ];

    __shared__ uint8_t matrixPivotValue;
    __shared__ uint8_t resultPivotValue;
    __shared__ uint8_t matrixCol[ SINGLE_BLOCK_SIZE ];
    __shared__ uint8_t resultCol[ SINGLE_BLOCK_SIZE];

	setup_tables(8);
	__syncthreads();

    if ( row < size )
    {
        if ( ty == 0 )
        {
            matrixPivotValue = matrix[ IDC2D(pivotIndex, col, size) ];
            resultPivotValue = result[ IDC2D(pivotIndex, col, size) ];
        }
        pivotCol[ty] = matrix[ IDC2D(row, pivotIndex, size) ];
        
        matrixCol[ty] = matrix[ IDC2D(row, col, size) ]; 
        resultCol[ty] = result[ IDC2D(row, col, size) ]; 
        __syncthreads();

		// substraction in GF
		// make the pivotCol become reduced echelon form
        if ( row != pivotIndex )
        {
			matrix[ IDC2D(row, col, size) ] = matrixCol[ty] ^ gf_mul(pivotCol[ty], matrixPivotValue);
			result[ IDC2D(row, col, size) ] = resultCol[ty] ^ gf_mul(pivotCol[ty], resultPivotValue);
        }
    }
}

//eliminate by column to make the pivot row become reduced echelon form
__global__ void eliminate_by_col(uint8_t *matrix, uint8_t *result, int pivotIndex, int size)
{
    int ty = threadIdx.y;

	int row = blockIdx.x;
	int col = blockDim.y * blockIdx.y + threadIdx.y;

    __shared__ uint8_t pivotRow[ SINGLE_BLOCK_SIZE ];

    __shared__ uint8_t matrixPivotValue;
    __shared__ uint8_t resultPivotValue;
    __shared__ uint8_t matrixCol[ SINGLE_BLOCK_SIZE ];
    __shared__ uint8_t resultCol[ SINGLE_BLOCK_SIZE];

	setup_tables(8);
	__syncthreads();

    if ( row < size )
    {
        if ( ty == 0 )
        {
            matrixPivotValue = matrix[ IDC2D(row, pivotIndex, size) ];
            resultPivotValue = result[ IDC2D(row, pivotIndex, size) ];
        }
        pivotRow[ty] = matrix[ IDC2D(pivotIndex, col, size) ];
        
        matrixCol[ty] = matrix[ IDC2D(row, col, size) ]; 
        resultCol[ty] = result[ IDC2D(row, col, size) ]; 
        __syncthreads();

		// substraction in GF
		// make the pivotRow become reduced echelon form
        if ( col != pivotIndex )
        {
			matrix[ IDC2D(row, col, size) ] = matrixCol[ty] ^ gf_mul(pivotRow[ty], matrixPivotValue);
			result[ IDC2D(row, col, size) ] = resultCol[ty] ^ gf_mul(pivotRow[ty], resultPivotValue);
        }
    }
}

//generate an identity matrix
__global__ void get_identity_matrix(uint8_t *result, int size)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

    if ( row == col )
	{
        result[ IDC2D(row, col, size) ] = 1;
	}
    else
	{
        result[ IDC2D(row, col, size) ] = 0;
	}
}

//find the pivot index in the given row/column
int get_pivot_index(uint8_t *vector, int index, int size)
{
    int pivotIndex = -1;
    int i = index;
    while( pivotIndex == -1 && i < size )
    {
        pivotIndex = (vector[i] > 0)? i: -1;        
        i++;
    }
    return pivotIndex;
}

#ifdef DEBUG
void show_squre_matrix_debug(uint8_t *matrix, int size)
{
	int i;
	int j;
	for(i=0; i<size; i++)
	{
		for(j=0; j<size; j++)
		{
			printf("%d ", matrix[i*size+j]);
		}
		printf("\n");
	}
		printf("\n");
}
#endif

// compute the inverse of a given matrix
// Gaussian/Gauss–Jordan elimination
extern "C"
void invert_matrix(uint8_t *matrix_dev, uint8_t *result_dev, int size)
{
	int row;
	int pivotIndex;
    uint8_t currentRow[size];
    int currentRowSize = size*sizeof(uint8_t);

    dim3 gimGrid( (int)(ceil( (float)size / SQUARE_BLOCK_SIZE)), (int)(ceil( (float)size / SQUARE_BLOCK_SIZE)) );
    dim3 gimBlock( min(size, SQUARE_BLOCK_SIZE), min(size, SQUARE_BLOCK_SIZE) );
    get_identity_matrix<<< gimGrid, gimBlock >>>(result_dev, size);
//	hipDeviceSynchronize();
	
	for( row = 0; row < size; row++ )
    {
		// check whether the leading coefficient of the current row is in the 'index'th column
		int index = row;
        hipMemcpy(currentRow, matrix_dev+row*size, currentRowSize, hipMemcpyDeviceToHost);
        pivotIndex = get_pivot_index(currentRow, index, size);
        if( pivotIndex != row )
		{
			dim3 scGrid(1, (int)(ceil( (float)size / SINGLE_BLOCK_SIZE )));
			dim3 scBlock(1, min(size, SINGLE_BLOCK_SIZE)); 
            switch_columns<<< scGrid, scBlock >>>(matrix_dev, result_dev, index, pivotIndex, size);
		}
		hipDeviceSynchronize();

		dim3 nprGrid(1, (int)(ceil( (float)size / SINGLE_BLOCK_SIZE )));
		dim3 nprBlock(1, min(size, SINGLE_BLOCK_SIZE)); 
    	// Normalize the pivot row
        normalize_pivot_row<<< nprGrid, nprBlock >>>(matrix_dev, result_dev, index, size);
//    	// Normalize the pivot column
//        normalize_pivot_col<<< nprGrid, linearBlock >>>(matrix_dev, result_dev, index, size);
		hipDeviceSynchronize();

		dim3 ebrGrid(size, (int)(ceil( (float)size / SINGLE_BLOCK_SIZE )));
		dim3 ebrBlock(1, min(size, SINGLE_BLOCK_SIZE)); 
        eliminate_by_row<<< ebrGrid, ebrBlock >>>(matrix_dev, result_dev, row, size);
		hipDeviceSynchronize();

#ifdef DEBUG
uint8_t matrix_host[size*size];
hipMemcpy(matrix_host, matrix_dev, size*size, hipMemcpyDeviceToHost);
printf("matrix:\n");
show_squre_matrix_debug(matrix_host, size);
uint8_t result_host[size*size];
hipMemcpy(result_host, result_dev, size*size, hipMemcpyDeviceToHost);
printf("result:\n");
show_squre_matrix_debug(result_host, size);
#endif
    }

}

__global__ void gen_encoding_matrix(uint8_t *encodingMatrix, int row, int col)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	setup_tables(8);
	__syncthreads();
	encodingMatrix[i*col + j] = gf_pow(j+1, i);
}

__global__ void encode_chunk(unsigned char *dataChunk, unsigned char *parityCoeff, unsigned char *codeChunk, int nativeBlockNum, int parityBlockNum, int chunkSize)
{
	matrix_mul(parityCoeff, dataChunk, codeChunk, parityBlockNum, nativeBlockNum, chunkSize);
/*
	int currentSize = chunkSize;
	for(int i=0; i<(int)(ceil((float)chunkSize/SINGLE_GRID_SIZE)); i++)
	{
		if(chunkSize-(i+1)*SINGLE_GRID_SIZE < 0)
		{
			currentSize = chunkSize - i*SINGLE_GRID_SIZE;
		}
		matrix_mul(parityCoeff, dataChunk+i*SINGLE_GRID_SIZE, codeChunk+i*SINGLE_GRID_SIZE, parityBlockNum, nativeBlockNum, currentSize);
	}
*/
}

__global__ void decode_chunk(unsigned char *dataChunk, unsigned char *parityCoeff, unsigned char *codeChunk, int nativeBlockNum, int parityBlockNum, int chunkSize)
{
	matrix_mul(parityCoeff, codeChunk, dataChunk, nativeBlockNum, nativeBlockNum, chunkSize);
/*
	int currentSize = chunkSize;
	for(int i=0; i<(int)(ceil((float)chunkSize/SINGLE_GRID_SIZE)); i++)
	{
		if(chunkSize-(i+1)*SINGLE_GRID_SIZE < 0)
		{
			currentSize = chunkSize - i*SINGLE_GRID_SIZE;
		}
		matrix_mul(parityCoeff, codeChunk+i*SINGLE_GRID_SIZE, dataChunk+i*SINGLE_GRID_SIZE, parityBlockNum, nativeBlockNum, currentSize);
	}
*/
}




